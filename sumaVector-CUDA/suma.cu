#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>



__global__ void vAdd(float* A, int num_elements, int factor_hilos, float* s){

	//__local__ float a = 0.0;
	//__shared__ float a;

	//if(threadIdx.x == 0) a = 0.0;
		//__syncthreads();

	//Posicion del thread
	int i = (blockIdx.x * blockDim.x + threadIdx.x);


	//printf("Hola desde el hilo %d, en el bloque %d y el hilo %d\n", i, blockIdx.x, threadIdx.x);

	if(i < factor_hilos*num_elements){


		atomicAdd(s, A[i%num_elements]);

		//atomicAdd(&a, 2);
		//A[i%num_elements] = A[i%num_elements] + 1;

	}

	//A[i%num_elements] = a;

	//s = a;

	//printf("%d", s[0]);


}



void fError(hipError_t err, int i){
	if(err != hipSuccess){
		printf("%d Ha ocurrido un error con codigo: %s\n", i, hipGetErrorString(err));
	}
}


int main(){

	//cudaSetDevice(1);

	int num_elements = 1024;
	int factor_hilos = 1;

	//Reservar espacio en memoria HOST


	float * h_A = (float*)malloc(num_elements * sizeof(float));


	if(h_A == NULL ){
		printf("Error al reservar memoria para los vectores HOST");
		exit(1);
	}


	float * h_sum = (float*)malloc(sizeof(float));
	h_sum[0] = 0;


	//Inicializar elementos de los vectores
	for(int i=0; i<num_elements; i++){
		h_A[i] = (float)i;

	}

	hipError_t err;

	float size = num_elements * sizeof(float);

	float * d_A = NULL;
	err = hipMalloc((void **)&d_A, size);
	fError(err,1);

	float * d_sum = NULL;
	err = hipMalloc((void **)&d_sum, sizeof(float));
	fError(err, 3);








	//Copiamos a GPU DEVICE
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_sum, h_sum, sizeof(float), hipMemcpyHostToDevice);



	int HilosPorBloque = 256;
	int BloquesPorGrid = (factor_hilos*num_elements + HilosPorBloque -1) / HilosPorBloque;


	hipError_t Err;

	//Lanzamos el kernel y medimos tiempos
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	vAdd<<<BloquesPorGrid, HilosPorBloque>>>(d_A, num_elements, factor_hilos, d_sum);
	Err = hipGetLastError();
	fError(Err,2);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float tiempo_reserva_host;
	hipEventElapsedTime(&tiempo_reserva_host, start, stop);


	printf("Tiempo de suma vectores DEVICE: %f\n", tiempo_reserva_host);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	//Copiamos a CPU el vector C
	err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);


	hipMemcpy(h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);




	/*for(int i=0; i<20; i++){
		printf("%f ", h_A[i]);
		//printf("\n");
	}*/

	printf("La suma es: %f", h_sum[0]);

}







